extern "C" {
#include "cudamem.cuh"
}

/* Pinned memory functions */

/**
 *    Pin allocated double vector
 */
void pin_double_vector(double *vector, long Nx) {
   if(hipHostRegister((void *) vector, Nx * sizeof(double), hipHostRegisterPortable) != hipSuccess) {
      fprintf(stderr, "Failed to pin memory for double vector.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/**
 *    Pin allocated complex vector
 */
void pin_complex_vector(hipDoubleComplex *vector, long Nx) {
   if(hipHostRegister((void *) vector, Nx * sizeof(hipDoubleComplex), hipHostRegisterPortable) != hipSuccess) {
      fprintf(stderr, "Failed to pin memory for complex vector.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/**
 *    Pin allocated double matrix
 */
void pin_double_matrix(double **matrix, long Nx, long Ny) {
   if(hipHostRegister((void *) matrix[0], Nx * Ny * sizeof(double), hipHostRegisterPortable) != hipSuccess) {
      fprintf(stderr, "Failed to pin memory for double matrix.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/**
 *    Pin allocated complex matrix
 */
void pin_complex_matrix(hipDoubleComplex **matrix, long Nx, long Ny) {
   if(hipHostRegister((void *) matrix[0], Nx * Ny * sizeof(hipDoubleComplex), hipHostRegisterPortable) != hipSuccess) {
      fprintf(stderr, "Failed to pin memory for complex matrix.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/**
 *    Pin allocated double tensor
 */
void pin_double_tensor(double ***tensor, long Nx, long Ny, long Nz) {
   if(hipHostRegister((void *) tensor[0][0], Nx * Ny * Nz * sizeof(double), hipHostRegisterPortable) != hipSuccess) {
      fprintf(stderr, "Failed to pin memory for double tensor.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/**
 *    Pin allocated complex tensor
 */
void pin_complex_tensor(hipDoubleComplex ***tensor, long Nx, long Ny, long Nz) {
   if(hipHostRegister((void *) tensor[0][0], Nx * Ny * Nz * sizeof(hipDoubleComplex), hipHostRegisterPortable) != hipSuccess) {
      fprintf(stderr, "Failed to pin memory for complex tensor.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/* Mapped memory functions */

/**
 *    Map pinned double matrix
 */
struct hipPitchedPtr map_double_matrix(double **matrix, long Nx, long Ny) {
   double *d_matrix;

   if(hipHostGetDevicePointer((void **) &d_matrix, matrix[0], 0) != hipSuccess) {
      fprintf(stderr, "Failed to get device pointer for double matrix in mapped memory.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return make_hipPitchedPtr(d_matrix, Ny * sizeof(double), Ny * sizeof(double), Nx);
}

/**
 *    Map pinned double matrix
 */
struct hipPitchedPtr map_double_matrix2(double *matrix, long Nx, long Ny) {
   double *d_matrix;

   if(hipHostGetDevicePointer((void **) &d_matrix, matrix, 0) != hipSuccess) {
      fprintf(stderr, "Failed to get device pointer for double matrix in mapped memory.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return make_hipPitchedPtr(d_matrix, Ny * sizeof(double), Ny * sizeof(double), Nx);
}

/**
 *    Map pinned complex matrix
 */
struct hipPitchedPtr map_complex_matrix(hipDoubleComplex **matrix, long Nx, long Ny) {
   hipDoubleComplex *d_matrix;

   if(hipHostGetDevicePointer((void **) &d_matrix, matrix[0], 0) != hipSuccess) {
      fprintf(stderr, "Failed to get device pointer for complex matrix in mapped memory.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return make_hipPitchedPtr(d_matrix, Ny * sizeof(hipDoubleComplex), Ny * sizeof(hipDoubleComplex), Nx);
}

/**
 *    Map pinned double tensor
 */
struct hipPitchedPtr map_double_tensor(double ***tensor, long Nx, long Ny, long Nz) {
   double *d_tensor;

   if(hipHostGetDevicePointer((void **) &d_tensor, tensor[0][0], 0) != hipSuccess) {
      fprintf(stderr, "Failed to get device pointer for double tensor in mapped memory.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return make_hipPitchedPtr(d_tensor, Nz * sizeof(double), Nz * sizeof(double), Ny);
}

/**
 *    Map pinned complex tensor
 */
struct hipPitchedPtr map_complex_tensor(hipDoubleComplex ***tensor, long Nx, long Ny, long Nz) {
   hipDoubleComplex *d_tensor;

   if(hipHostGetDevicePointer((void **) &d_tensor, tensor[0][0], 0) != hipSuccess) {
      fprintf(stderr, "Failed to get device pointer for complex tensor in mapped memory.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return make_hipPitchedPtr(d_tensor, Nz * sizeof(hipDoubleComplex), Nz * sizeof(hipDoubleComplex), Ny);
}

/* Pinned/mapped memory release */

/**
 *    Free pinned/mapped double vector
 */
void free_pinned_double_vector(double *vector) {
   if (hipHostUnregister(vector) != hipSuccess) {
      fprintf(stderr, "Failed to unregister pinned memory for double vector.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   //free_double_vector(vector);
}

/**
 *    Free pinned/mapped complex vector
 */
void free_pinned_complex_vector(hipDoubleComplex *vector) {
   if (hipHostUnregister(vector) != hipSuccess) {
      fprintf(stderr, "Failed to unregister pinned memory for complex vector.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   //free_complex_vector(vector);
}

/**
 *    Free pinned/mapped double matrix
 */
void free_pinned_double_matrix(double **matrix) {
   if (hipHostUnregister(matrix[0]) != hipSuccess) {
      fprintf(stderr, "Failed to unregister pinned memory for double matrix.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   //free_double_matrix(matrix);
}

/**
 *    Free pinned/mapped complex matrix
 */
void free_pinned_complex_matrix(hipDoubleComplex **matrix) {
   if (hipHostUnregister(matrix[0]) != hipSuccess) {
      fprintf(stderr, "Failed to unregister pinned memory for complex matrix.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   //free_complex_matrix(matrix);
}

/**
 *    Free pinned/mapped double tensor
 */
void free_pinned_double_tensor(double ***tensor) {
   if (hipHostUnregister(tensor[0][0]) != hipSuccess) {
      fprintf(stderr, "Failed to unregister pinned memory for double tensor.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   //free_double_tensor(tensor);
}

/**
 *    Free pinned/mapped complex tensor
 */
void free_pinned_complex_tensor(hipDoubleComplex ***tensor) {
   if (hipHostUnregister(tensor[0][0]) != hipSuccess) {
      fprintf(stderr, "Failed to unregister pinned memory for complex tensor.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   //free_complex_tensor(tensor);
}

/* CUDA memory functions */

/**
 *    Double vector allocation on CUDA device
 */
double *alloc_double_vector_device(long Nx) {
   double *vector;

   if(hipMalloc((void**) &vector, Nx * sizeof(double)) != hipSuccess) {
      fprintf(stderr, "Failed to allocate device memory for the CUDA double vector.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return vector;
}

/**
 *    Complex vector allocation on CUDA device
 */
hipDoubleComplex *alloc_complex_vector_device(long Nx) {
   hipDoubleComplex *vector;

   if(hipMalloc((void**) &vector, Nx * sizeof(hipDoubleComplex)) != hipSuccess) {
      fprintf(stderr, "Failed to allocate device memory for the CUDA complex vector.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return vector;
}

/**
 *    Double matrix allocation on CUDA device
 */
struct hipPitchedPtr alloc_double_matrix_device(long Nx, long Ny) {
   struct hipPitchedPtr matrix;

   if(hipMalloc3D(&matrix, make_hipExtent(Ny * sizeof(double), Nx, 1)) != hipSuccess) {
      fprintf(stderr, "Failed to allocate device memory for the CUDA double matrix.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return matrix;
}

/**
 *    Complex matrix allocation on CUDA device
 */
struct hipPitchedPtr alloc_complex_matrix_device(long Nx, long Ny) {
   struct hipPitchedPtr matrix;

   if(hipMalloc3D(&matrix, make_hipExtent(Ny * sizeof(hipDoubleComplex), Nx, 1)) != hipSuccess) {
      fprintf(stderr, "Failed to allocate device memory for the CUDA complex matrix.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return matrix;
}

/**
 *    Double tensor allocation on CUDA device
 */
struct hipPitchedPtr alloc_double_tensor_device(long Nx, long Ny, long Nz) {
   struct hipPitchedPtr tensor;

   if(hipMalloc3D(&tensor, make_hipExtent(Nz * sizeof(double), Ny, Nx)) != hipSuccess) {
      fprintf(stderr, "Failed to allocate device memory for the CUDA double tensor.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return tensor;
}

/**
 *    Complex tensor allocation on CUDA device
 */
struct hipPitchedPtr alloc_complex_tensor_device(long Nx, long Ny, long Nz) {
   struct hipPitchedPtr tensor;

   if(hipMalloc3D(&tensor, make_hipExtent(Nz * sizeof(hipDoubleComplex), Ny, Nx)) != hipSuccess) {
      fprintf(stderr, "Failed to allocate device memory for the CUDA complex tensor.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   return tensor;
}

/**
 *    Free double vector on CUDA device
 */
void free_double_vector_device(double *vector) {
   if (hipFree(vector) != hipSuccess) {
      fprintf(stderr, "Failed to free device memory for double vector.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/**
 *    Free complex vector on CUDA device
 */
void free_complex_vector_device(hipDoubleComplex *vector) {
   if (hipFree(vector) != hipSuccess) {
      fprintf(stderr, "Failed to free device memory for complex vector.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/**
 *    Free double matrix on CUDA device
 */
void free_double_matrix_device(struct hipPitchedPtr matrix) {
   if (hipFree(matrix.ptr) != hipSuccess) {
      fprintf(stderr, "Failed to free device memory for double matrix.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/**
 *    Free complex matrix on CUDA device
 */
void free_complex_matrix_device(struct hipPitchedPtr matrix) {
   if (hipFree(matrix.ptr) != hipSuccess) {
      fprintf(stderr, "Failed to free device memory for complex matrix.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/**
 *    Free double tensor on CUDA device
 */
void free_double_tensor_device(struct hipPitchedPtr tensor) {
   if (hipFree(tensor.ptr) != hipSuccess) {
      fprintf(stderr, "Failed to free device memory for double tensor.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

/**
 *    Free complex tensor on CUDA device
 */
void free_complex_tensor_device(struct hipPitchedPtr tensor) {
   if (hipFree(tensor.ptr) != hipSuccess) {
      fprintf(stderr, "Failed to free device memory for complex tensor.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}




static char *d_pool;
static size_t pool_size = 0;
static size_t offset = 0;
void init_mem_device(size_t size) {

   if(hipMalloc((void**) &d_pool, size) != hipSuccess) {
      fprintf(stderr, "Failed to allocate %zu of device memory.\n", size);
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   pool_size = size;
   offset = 0;
}

void *alloc_mem_device(size_t size) {
   void *d_ptr;

   d_ptr = d_pool + offset;
   offset += size;

   if (offset > pool_size) {
      fprintf(stderr, "Failed to allocate %zu of device memory.\n", size);
      exit(EXIT_FAILURE);
   }

   return d_ptr;
}

void reset_mem_device() {
   offset = 0;
}

void free_mem_device() {
   if (hipFree(d_pool) != hipSuccess) {
      fprintf(stderr, "Failed to free device memory.\n");
      hipError_t error = hipGetLastError();
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }

   offset = 0;
}
